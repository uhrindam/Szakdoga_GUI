#include "hip/hip_runtime.h"
#include "slicCUDA.h"

__device__ int *d_clusters;								//cols * rows
__device__ float *d_distances;							//cols * rows
__device__ float *d_centers;							//centersLength * 5
__device__ int *d_center_counts;						//centersLength
__device__ uchar3 *d_colors;							//cols * rows
__device__ int *d_neighbors;							//centerlength * 8

slicCUDA::slicCUDA(){}

slicCUDA::~slicCUDA(){}

__device__ float compute_dist(int ci, int y, int x, uchar3 colour, float *d_centers, int pitch, int d_step)
{
	//sz�nt�vols�g
	float dc = sqrt(pow(d_centers[ci *pitch + 0] - colour.x, 2) + pow(d_centers[ci *pitch + 1] - colour.y, 2)
		+ pow(d_centers[ci *pitch + 2] - colour.z, 2));
	//euklideszi t�vols�g
	float ds = sqrt(pow(d_centers[ci *pitch + 3] - x, 2) + pow(d_centers[ci *pitch + 4] - y, 2));

	return sqrt(pow(dc / nc, 2) + pow(ds / d_step, 2));
}

__global__ void compute(int d_cols, int d_rows, int d_step, int d_centersLength, int *d_clusters, float *d_distances,
	float *d_centers, int *d_center_counts, uchar3 *d_colors, int pitch)
{
	int clusterIDX = blockIdx.x * blockDim.x + threadIdx.x;

	if (clusterIDX < d_centersLength)
	{
		for (int pixelY = d_centers[clusterIDX *pitch + 3] - (d_step*1.5); pixelY < d_centers[clusterIDX *pitch + 3] + (d_step*1.5); pixelY++)
		{
			for (int pixelX = d_centers[clusterIDX *pitch + 4] - (d_step*1.5); pixelX < d_centers[clusterIDX *pitch + 4] + (d_step*1.5); pixelX++)
			{
				if (pixelX >= 0 && pixelX < d_rows && pixelY >= 0 && pixelY < d_cols)
				{
					uchar3 colour = d_colors[d_rows*pixelY + pixelX];
					float distance = compute_dist(clusterIDX, pixelX, pixelY, colour, d_centers, pitch, d_step);
					if (distance < d_distances[d_rows*pixelY + pixelX])
					{
						d_distances[d_rows*pixelY + pixelX] = distance;
						d_clusters[d_rows*pixelY + pixelX] = clusterIDX;
					}
				}
			}
		}
		//a centroidok alaphelyzetbe �ll�t�sa
		d_centers[clusterIDX *pitch + 0] = 0;
		d_centers[clusterIDX *pitch + 1] = 0;
		d_centers[clusterIDX *pitch + 2] = 0;
		d_centers[clusterIDX *pitch + 3] = 0;
		d_centers[clusterIDX *pitch + 4] = 0;
		d_center_counts[clusterIDX] = 0;
	}

}

__global__ void compute1(int d_cols, int d_rows, int *d_clusters, float *d_distances,
	float *d_centers, int *d_center_counts, uchar3 *d_colors, int pitch)
{
	int idIn1D = blockIdx.x * blockDim.x + threadIdx.x;
	if (idIn1D < d_cols*d_rows)
	{
		d_distances[idIn1D] = FLT_MAX;

		int whichCluster = d_clusters[idIn1D];
		atomicAdd(&d_centers[whichCluster*pitch + 0], d_colors[idIn1D].x);
		atomicAdd(&d_centers[whichCluster*pitch + 1], d_colors[idIn1D].y);
		atomicAdd(&d_centers[whichCluster*pitch + 2], d_colors[idIn1D].z);
		atomicAdd(&d_centers[whichCluster*pitch + 3], idIn1D / d_rows);
		atomicAdd(&d_centers[whichCluster*pitch + 4], idIn1D % d_rows);

		atomicAdd(&d_center_counts[whichCluster], 1);
	}
}

__global__ void compute2(int d_centersLength, float *d_centers, int *d_center_counts, int pitch)
{
	int idIn1D = blockIdx.x * blockDim.x + threadIdx.x;
	if (idIn1D < d_centersLength)
	{
		d_centers[idIn1D*pitch + 0] = (int)(d_centers[idIn1D*pitch + 0] / d_center_counts[idIn1D]);
		d_centers[idIn1D*pitch + 1] = (int)(d_centers[idIn1D*pitch + 1] / d_center_counts[idIn1D]);
		d_centers[idIn1D*pitch + 2] = (int)(d_centers[idIn1D*pitch + 2] / d_center_counts[idIn1D]);
		d_centers[idIn1D*pitch + 3] = (int)(d_centers[idIn1D*pitch + 3] / d_center_counts[idIn1D]);
		d_centers[idIn1D*pitch + 4] = (int)(d_centers[idIn1D*pitch + 4] / d_center_counts[idIn1D]);
	}
}

float slicCUDA::colorDistance(uchar3 actuallPixel, uchar3 neighborPixel)
{
	float dc = sqrt(pow(actuallPixel.x - neighborPixel.x, 2) + pow(actuallPixel.y - neighborPixel.y, 2)
		+ pow(actuallPixel.z - neighborPixel.z, 2));
	return dc;
}

void slicCUDA::neighborMerge()
{
	const int dx8[numberOfNeighbors] = { -1, -1,  0,  1, 1, 1, 0, -1 };
	const int dy8[numberOfNeighbors] = { 0, -1, -1, -1, 0, 1, 1,  1 };

	for (int i = 0; i < centersLength; i++)
	{
		uchar3 actuallCluster;
		actuallCluster.x = centers[i * 5];
		actuallCluster.y = centers[i * 5 + 1];
		actuallCluster.z = centers[i * 5 + 2];

		int clusterRow = i / centersRowPieces;
		int clusterCol = i % centersRowPieces;

		for (int j = 0; j < numberOfNeighbors; j++)
		{
			if (clusterCol + dy8[j] >= 0 && clusterCol + dy8[j] < centersRowPieces
				&& clusterRow + dx8[j] >= 0 && clusterRow + dx8[j] < centersColPieces)
			{
				uchar3 neighborPixel;
				neighborPixel.x = centers[(centersRowPieces* (clusterRow + dx8[j]) + (clusterCol + dy8[j])) * 5 + 0];
				neighborPixel.y = centers[(centersRowPieces* (clusterRow + dx8[j]) + (clusterCol + dy8[j])) * 5 + 1];
				neighborPixel.z = centers[(centersRowPieces* (clusterRow + dx8[j]) + (clusterCol + dy8[j])) * 5 + 2];

				if (centersRowPieces * clusterRow + clusterCol < centersRowPieces * (clusterRow + dx8[j]) + (clusterCol + dy8[j]) &&
					colorDistance(actuallCluster, neighborPixel) < maxColorDistance)
				{
					neighbors[(centersRowPieces * clusterRow + clusterCol) * numberOfNeighbors + j] = centersRowPieces * (clusterRow + dx8[j]) + (clusterCol + dy8[j]);
				}
			}
		}
	}

	int2 *changes = new int2[centersLength];
	for (int i = 0; i < centersLength; i++)
	{
		changes[i].x = i;
		changes[i].y = -1;
	}

	for (int i = 0; i < centersLength; i++)
	{
		for (int j = 0; j < numberOfNeighbors; j++)
		{
			int cluster = neighbors[i * numberOfNeighbors + j];
			if (cluster != -1)
			{
				int neighborIDX = changes[cluster].y;
				int clusterIDX = i;
				while (neighborIDX != -1)
				{
					neighborIDX = changes[neighborIDX].y;
					if (neighborIDX != -1)
						clusterIDX = changes[neighborIDX].x;
				}
				if (changes[clusterIDX].y != -1)
					changes[cluster].y = changes[clusterIDX].y;
				else
					changes[cluster].y = clusterIDX;
			}
		}
	}

	for (int i = 0; i < cols*rows; i++)
	{
		if (changes[clusters[i]].y != -1)
		{
			clusters[i] = changes[clusters[i]].y;
		}
	}
}

void slicCUDA::initData(Mat image)
{
	cols = image.cols;
	rows = image.rows;
	step = (sqrt((cols * rows) / (double)numberofSuperpixels));

	clusters = new int[cols*rows];
	distances = new float[cols*rows];
	for (int i = 0; i < cols*rows; i++)
	{
		clusters[i] = -1;
		distances[i] = FLT_MAX;
	}

	//Ez az�rt kell mert el�re nem tudom, hogy h�ny eleme lesz a centers-nek, ez�rt el�sz�r egy vectorhoz adogatom hozz� az elemeket
	// majd k�s�bb l�trehozom a t�mb�t annyi elemmel, ah�ny eleme van a seg�dvectornak, majd �tm�solom az adatokat.
	centersColPieces = 0;
	centersRowPieces = 0;
	vector<vector<float> > h_centers;
	for (int i = step; i < cols - step / 2; i += step) {
		for (int j = step; j < rows - step / 2; j += step) {
			vector<float> center;
			Vec3b colour = image.at<Vec3b>(j, i);

			center.push_back(colour.val[0]);
			center.push_back(colour.val[1]);
			center.push_back(colour.val[2]);
			center.push_back(i);
			center.push_back(j);

			h_centers.push_back(center);
		}
		centersColPieces++;
	}

	centersLength = h_centers.size();
	centersRowPieces = centersLength / centersColPieces;

	centers = new float[centersLength * 5];
	center_counts = new int[centersLength];
	neighbors = new int[centersLength * numberOfNeighbors];

	int idx = 0;
	for (int i = 0; i < centersLength; i++)
	{
		for (int j = 0; j < 5; j++)
		{
			centers[idx] = h_centers[i][j];
			idx++;
		}
		for (int j = 0; j < numberOfNeighbors; j++)
		{
			neighbors[i * numberOfNeighbors + j] = -1;
		}
		center_counts[i] = 0;
	}

	//Bej�rom a k�pet, majd minden pixel sz�n�t (3 �rt�k) elmentem egy uchar3 v�ltoz�ba
	colors = new uchar3[rows*cols];
	for (int i = 0; i < cols; i++)
	{
		for (int j = 0; j < rows; j++)
		{
			Vec3b colour = image.at<Vec3b>(j, i);
			colors[i * rows + j] = make_uchar3(colour.val[0], colour.val[1], colour.val[2]);
		}
	}
}



void slicCUDA::dataCopy()
{
	hipMalloc((void**)&d_clusters, sizeof(int)*rows*cols);
	hipMemcpy(d_clusters, clusters, sizeof(int)*rows*cols, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_distances, sizeof(float)*rows*cols);
	hipMemcpy(d_distances, distances, sizeof(float)*rows*cols, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_centers, sizeof(float)*centersLength * 5);
	hipMemcpy(d_centers, centers, sizeof(float)*centersLength * 5, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_center_counts, sizeof(int)*centersLength);
	hipMemcpy(d_center_counts, center_counts, sizeof(int)*centersLength, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_colors, sizeof(uchar3)*rows*cols);
	hipMemcpy(d_colors, colors, sizeof(uchar3)*rows*cols, hipMemcpyHostToDevice);
}

void slicCUDA::dataFree()
{
	hipFree(d_clusters);
	hipFree(d_distances);
	hipFree(d_centers);
	hipFree(d_center_counts);
	hipFree(d_colors);
}

void slicCUDA::colour_with_cluster_means(Mat image) {
	cout << "FILL" << endl;

	for (int i = 0; i < image.cols; i++) {
		for (int j = 0; j < image.rows; j++) {
			int idx = clusters[i*image.rows + j];
			Vec3b ncolour = image.at<Vec3b>(j, i);

			ncolour.val[0] = centers[idx * 5 + 0];
			ncolour.val[1] = centers[idx * 5 + 1];
			ncolour.val[2] = centers[idx * 5 + 2];

			image.at<Vec3b>(j, i) = ncolour;
		}
	}
}

void slicCUDA::startKernels()
{
	int howManyBlocks = centersLength / maxThreadinoneBlock;
	int threadsPerBlock = (centersLength / howManyBlocks) + 1;

	int howManyBlocks2 = rows*cols / maxThreadinoneBlock;
	int threadsPerBlock2 = (rows*cols / howManyBlocks2) + 1;
	for (int i = 0; i < iteration; i++)
	{
		dataCopy();
		compute << <howManyBlocks, threadsPerBlock >> > (cols, rows, step, centersLength, d_clusters, d_distances, d_centers, d_center_counts, d_colors, 5);
		compute1 << <howManyBlocks2, threadsPerBlock2 >> > (cols, rows, d_clusters, d_distances, d_centers, d_center_counts, d_colors, 5);
		compute2 << <howManyBlocks, threadsPerBlock >> > (centersLength, d_centers, d_center_counts, 5);

		hipMemcpy(distances, d_distances, sizeof(float)*rows*cols, hipMemcpyDeviceToHost);
		hipMemcpy(clusters, d_clusters, sizeof(int)*rows*cols, hipMemcpyDeviceToHost);
		hipMemcpy(centers, d_centers, sizeof(int)*centersLength * 5, hipMemcpyDeviceToHost);
		hipMemcpy(center_counts, d_center_counts, sizeof(int)*centersLength, hipMemcpyDeviceToHost);

		dataFree();
	}
}