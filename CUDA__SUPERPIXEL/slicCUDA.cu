#include "hip/hip_runtime.h"
#include "slicCUDA.h"

__device__ int *d_clusters;								//cols * rows
__device__ float *d_distances;							//cols * rows
__device__ float *d_centers;							//centersLength * 5
__device__ int *d_center_counts;						//centersLength
__device__ uchar3 *d_colors;							//cols * rows

slicCUDA::slicCUDA() {}

slicCUDA::~slicCUDA() {}

__device__ float compute_dist(int ci, int y, int x, uchar3 colour, float *d_centers, int pitch, int d_step)
{
	//sz�nt�vols�g
	float dc = sqrt(pow(d_centers[ci *pitch + 0] - colour.x, 2) + pow(d_centers[ci *pitch + 1] - colour.y, 2)
		+ pow(d_centers[ci *pitch + 2] - colour.z, 2));
	//euklideszi t�vols�g
	float ds = sqrt(pow(d_centers[ci *pitch + 3] - x, 2) + pow(d_centers[ci *pitch + 4] - y, 2));

	return sqrt(pow(dc / nc, 2) + pow(ds / d_step, 2));
}

//l�p�ssz�m: centroidok sz�ma
//Itt rendelem a pixeleket az egyes clusterekhez sz�n, valamint euklideszi t�vols�g szerint
__global__ void orderingPixelsForClustersKernel(int d_cols, int d_rows, int d_step, int d_centersLength,
	int *d_clusters, float *d_distances, float *d_centers, int *d_center_counts, uchar3 *d_colors, int pitch)
{
	int clusterIDX = blockIdx.x * blockDim.x + threadIdx.x;
	//mivel nem tudok pontosan annyi sz�lat ind�tani ah�ny clusterem van, 
	//ez�rt megvizsg�lom, hogy az adott clusterindex m�g l�tezik-e
	if (clusterIDX < d_centersLength)
	{
		//Bej�rom az adott cluster "step" sugar� k�rnyezet�t
		//Az itt tal�lhat� pixelek mindegyik�re megn�zem, hogy az aktu�lisan vizsg�lt
		//centroid van-e hozz� a legk�zelebb, �s ha igen, akkor be�ll�tom a megfelel� adatokat
		//Ez a k�t egybe�gyazott forciklus miatt hossz�nak t�nik, de alapvet�en ez kev�s l�p�sb�l �ll
		for (int pixelY = d_centers[clusterIDX *pitch + 3] - (d_step*1.5); pixelY <
			d_centers[clusterIDX *pitch + 3] + (d_step*1.5); pixelY++)
		{
			for (int pixelX = d_centers[clusterIDX *pitch + 4] - (d_step*1.5); pixelX <
				d_centers[clusterIDX *pitch + 4] + (d_step*1.5); pixelX++)
			{
				//Ellen�rz�m a hat�rokat
				if (pixelX >= 0 && pixelX < d_rows && pixelY >= 0 && pixelY < d_cols)
				{
					uchar3 colour = d_colors[d_rows*pixelY + pixelX];
					float distance = compute_dist(clusterIDX, pixelX, pixelY, colour, d_centers, pitch, d_step);
					//ha a t�vols�g kisebb mint az eddig mentett (a default az FLT_MAX) akkor be�ll�tom 
					//az aktu�lis centroidot a legk�zelebbinek
					if (distance < d_distances[d_rows*pixelY + pixelX])
					{
						d_distances[d_rows*pixelY + pixelX] = distance;
						d_clusters[d_rows*pixelY + pixelX] = clusterIDX;
					}
				}
			}
		}

		//a centroidok alaphelyzetbe �ll�t�sa
		d_centers[clusterIDX *pitch + 0] = 0;
		d_centers[clusterIDX *pitch + 1] = 0;
		d_centers[clusterIDX *pitch + 2] = 0;
		d_centers[clusterIDX *pitch + 3] = 0;
		d_centers[clusterIDX *pitch + 4] = 0;
		d_center_counts[clusterIDX] = 0;
	}

}

//l�p�ssz�m: pixelek sz�ma
//Itt �sszegzem a kor�bban kapott eredm�nyeket.
__global__ void clusterValuesSumByPixelsKernel(int d_cols, int d_rows, int *d_clusters, float *d_distances,
	float *d_centers, int *d_center_counts, uchar3 *d_colors, int pitch)
{
	int idIn1D = blockIdx.x * blockDim.x + threadIdx.x;
	if (idIn1D < d_cols*d_rows)
	{
		//Alaphelyzetbe �ll�tom a t�vols�g�rt�keket minden piyel eset�ben
		d_distances[idIn1D] = FLT_MAX;

		//Megkeresem, hogy az adott pixel melyik centroidhoz tartozik.
		//amint ez megvan, �sszegzem ezeket az �rt�keket, atomi m�velettel, ugyanis
		//el�fordulhat hogy egy centroidhoz tartoz� t�mb�rt�ket egyszerre t�bb pixelsz�l is szeretne �rni
		//majd n�velem a centroidhoz tartoz� pixelek sz�m�t
		int whichCluster = d_clusters[idIn1D];
		atomicAdd(&d_centers[whichCluster*pitch + 0], d_colors[idIn1D].x);
		atomicAdd(&d_centers[whichCluster*pitch + 1], d_colors[idIn1D].y);
		atomicAdd(&d_centers[whichCluster*pitch + 2], d_colors[idIn1D].z);
		atomicAdd(&d_centers[whichCluster*pitch + 3], idIn1D / d_rows);
		atomicAdd(&d_centers[whichCluster*pitch + 4], idIn1D % d_rows);

		atomicAdd(&d_center_counts[whichCluster], 1);
	}
}

//l�p�ssz�m: centroidok sz�ma
//Az �sszegzett centroid�rt�keket elosztom a centroidhoz tartoz� pixelek darabsz�m�val.
__global__ void computeCorrectCentroidValuesKernel(int d_centersLength, float *d_centers, int *d_center_counts, int pitch)
{
	int idIn1D = blockIdx.x * blockDim.x + threadIdx.x;
	if (idIn1D < d_centersLength)
	{
		d_centers[idIn1D*pitch + 0] = (int)(d_centers[idIn1D*pitch + 0] / d_center_counts[idIn1D]);
		d_centers[idIn1D*pitch + 1] = (int)(d_centers[idIn1D*pitch + 1] / d_center_counts[idIn1D]);
		d_centers[idIn1D*pitch + 2] = (int)(d_centers[idIn1D*pitch + 2] / d_center_counts[idIn1D]);
		d_centers[idIn1D*pitch + 3] = (int)(d_centers[idIn1D*pitch + 3] / d_center_counts[idIn1D]);
		d_centers[idIn1D*pitch + 4] = (int)(d_centers[idIn1D*pitch + 4] / d_center_counts[idIn1D]);
	}
}

//A szomsz�dok �sszevon�sakor haszn�lt sz�nt�vols�g sz�m�t� f�ggv�ny
float slicCUDA::colorDistance(uchar3 actuallPixel, uchar3 neighborPixel)
{
	float dc = sqrt(pow(actuallPixel.x - neighborPixel.x, 2) + pow(actuallPixel.y - neighborPixel.y, 2)
		+ pow(actuallPixel.z - neighborPixel.z, 2));
	return dc;
}

//Itt ker�lnek �sszevon�sra a szomsz�dos hasonl� sz�n� szegmensek
void slicCUDA::neighborMerge()
{
	const int dx8[numberOfNeighbors] = { -1, -1,  0,  1, 1, 1, 0, -1 };
	const int dy8[numberOfNeighbors] = { 0, -1, -1, -1, 0, 1, 1,  1 };

	for (int i = 0; i < centersLength; i++)
	{
		//kimentem az aktu�lis centroid �rt�keit
		uchar3 actuallCluster;
		actuallCluster.x = centers[i * 5];
		actuallCluster.y = centers[i * 5 + 1];
		actuallCluster.z = centers[i * 5 + 2];

		int clusterRow = i / centersRowPieces;
		int clusterCol = i % centersRowPieces;

		//megn�zem az aktu�lis centroid szomsz�djait
		for (int j = 0; j < numberOfNeighbors; j++)
		{
			//ellen�rz�m a hat�rokat
			if (clusterCol + dy8[j] >= 0 && clusterCol + dy8[j] < centersRowPieces
				&& clusterRow + dx8[j] >= 0 && clusterRow + dx8[j] < centersColPieces)
			{
				//kimentem a szomsz�dos centroid adatait
				uchar3 neighborPixel;
				neighborPixel.x = centers[(centersRowPieces* (clusterRow + dx8[j]) + (clusterCol + dy8[j])) * 5 + 0];
				neighborPixel.y = centers[(centersRowPieces* (clusterRow + dx8[j]) + (clusterCol + dy8[j])) * 5 + 1];
				neighborPixel.z = centers[(centersRowPieces* (clusterRow + dx8[j]) + (clusterCol + dy8[j])) * 5 + 2];

				//ha az aktu�lis centroid sorsz�ma kisebb mint a szomsz�d sorsz�ma, valamint a sz�nt�vols�g a megengedett hat�ron
				//bel�l van, akkor felveszem az �sszevonand� sz�msz�dok k�z�.
				if (centersRowPieces * clusterRow + clusterCol < centersRowPieces * (clusterRow + dx8[j]) +
					(clusterCol + dy8[j]) && colorDistance(actuallCluster, neighborPixel) < maxColorDistance)
				{
					neighbors[(centersRowPieces * clusterRow + clusterCol) * numberOfNeighbors + j] =
						centersRowPieces * (clusterRow + dx8[j]) + (clusterCol + dy8[j]);
				}
			}
		}
	}

	//inicializ�lok egy seg�dt�mb�t, amelyben el fogom t�rolni hogy az egyes centroidokat melyik m�sik centroiddal kell �sszevonni
	int2 *changes = new int2[centersLength];
	for (int i = 0; i < centersLength; i++)
	{
		changes[i].x = i;
		changes[i].y = -1;
	}

	//Az itt k�vetkez� k�dr�sznek az a l�nyege, hogy az �sszevonand� centroidokat �sszel�ncolom �gy, hogy az egym�shoz k�zel l�v�
	//megengedett sz�nt�vols�g� centroidok �ssze legyenek vonva, annak elker�l�se v�gett, hogy esetleg egy centroid egy olyan m�sik
	//centroiddal legyen �sszevonva, amelyet m�r �sszevontam egy m�sikkal.
	//P�ld�ul: a k�p sz�l�n tal�lhat� egy feh�r keret, amelyen 500 centroid helyezkedik el. Ezeket p�ross�val is �sszevonhatn�m, de
	//ehelyett mind az 500-at egy centroidd� alak�tom, �s egyben kezelem az eg�szet.
	for (int i = 0; i < centersLength; i++)
	{
		for (int j = 0; j < numberOfNeighbors; j++)
		{
			//kimentem, hogy az adott szomsz�d az melyik centroid
			int cluster = neighbors[i * numberOfNeighbors + j];
			if (cluster != -1)
			{
				//kimentem, hogy az adott centroidot melyik m�sikkal kell �sszevonni
				int neighborIDX = changes[cluster].y;
				int clusterIDX = i;
				//Addig megyek v�gig az �sszevonand�kon am�g el nem �rek egy oylan centroidig, amit m�r nem kell m�sikkal �sszevonni
				//(Garant�ltan van olyan centroid a l�nc v�g�n amelyet nem kell m�sikkal �sszevonni, annak k�sz�nhet�en, hogy 
				//csak akkor mentem el szomsz�dk�nt az adott centroidot ha annak sorsz�ma nagyobb mint az aktu�lisan vizsg�lt)
				while (neighborIDX != -1)
				{
					neighborIDX = changes[neighborIDX].y;
					if (neighborIDX != -1)
						clusterIDX = changes[neighborIDX].x;
				}
				if (changes[clusterIDX].y != -1)
					changes[cluster].y = changes[clusterIDX].y;
				else
					changes[cluster].y = clusterIDX;
			}
		}
	}

	//V�g�l kimentem minden pixel eset�ben, hogy melyik az �j centroid amhez mostant�l tartoznak.
	for (int i = 0; i < cols*rows; i++)
	{
		if (changes[clusters[i]].y != -1)
		{
			clusters[i] = changes[clusters[i]].y;
		}
	}
}

void slicCUDA::initData(Mat image)
{
	cols = image.cols;
	rows = image.rows;
	step = (sqrt((cols * rows) / (double)numberofSuperpixels));

	//felt�lt�s default adatokkal
	clusters = new int[cols*rows];
	distances = new float[cols*rows];
	for (int i = 0; i < cols*rows; i++)
	{
		clusters[i] = -1;
		distances[i] = FLT_MAX;
	}

	centersColPieces = 0;
	centersRowPieces = 0;
	//Ez az�rt kell mert el�re nem tudom, hogy h�ny eleme lesz a centers-nek, ez�rt el�sz�r egy vectorhoz adogatom hozz� az elemeket
	// majd k�s�bb l�trehozom a t�mb�t annyi elemmel, ah�ny eleme van a seg�dvectornak, majd �tm�solom az adatokat.
	vector<vector<float> > h_centers;
	for (int i = step; i < cols - step / 2; i += step) {
		for (int j = step; j < rows - step / 2; j += step) {
			vector<float> center;
			Vec3b colour = image.at<Vec3b>(j, i);

			center.push_back(colour.val[0]);
			center.push_back(colour.val[1]);
			center.push_back(colour.val[2]);
			center.push_back(i);
			center.push_back(j);

			h_centers.push_back(center);
		}
		centersColPieces++;
	}

	centersLength = h_centers.size();
	centersRowPieces = centersLength / centersColPieces;

	centers = new float[centersLength * 5];
	center_counts = new int[centersLength];
	neighbors = new int[centersLength * numberOfNeighbors];
	//A centers �gy t�rolja az adatokat, hogy egy pixelhez let�rolja annak x, �s y poz�ci�j�t a k�pen, valamint
	//az adott pixel R, G �s B sz�nkomponenseit
	//A szomsz�dt�mb eset�ben pedig felt�lt�m a 8 szomsz�dot jelz� �rt�ket default adattal.
	int idx = 0;
	for (int i = 0; i < centersLength; i++)
	{
		for (int j = 0; j < 5; j++)
		{
			centers[idx] = h_centers[i][j];
			idx++;
		}
		for (int j = 0; j < numberOfNeighbors; j++)
		{
			neighbors[i * numberOfNeighbors + j] = -1;
		}
		center_counts[i] = 0;
	}

	//Bej�rom a k�pet, majd minden pixel sz�n�t (3 �rt�k) elmentem egy uchar3 v�ltoz�ba
	colors = new uchar3[rows*cols];
	for (int i = 0; i < cols; i++)
	{
		for (int j = 0; j < rows; j++)
		{
			Vec3b colour = image.at<Vec3b>(j, i);
			colors[i * rows + j] = make_uchar3(colour.val[0], colour.val[1], colour.val[2]);
		}
	}
}


//A kerneleken haszn�land� t�mb�k mem�riafoglal�sa, majd �tm�sol�sa
void slicCUDA::dataCopy()
{
	hipMalloc((void**)&d_clusters, sizeof(int)*rows*cols);
	hipMemcpy(d_clusters, clusters, sizeof(int)*rows*cols, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_distances, sizeof(float)*rows*cols);
	hipMemcpy(d_distances, distances, sizeof(float)*rows*cols, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_centers, sizeof(float)*centersLength * 5);
	hipMemcpy(d_centers, centers, sizeof(float)*centersLength * 5, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_center_counts, sizeof(int)*centersLength);
	hipMemcpy(d_center_counts, center_counts, sizeof(int)*centersLength, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_colors, sizeof(uchar3)*rows*cols);
	hipMemcpy(d_colors, colors, sizeof(uchar3)*rows*cols, hipMemcpyHostToDevice);
}

//A device t�mb�k felszabad�t�sa
void slicCUDA::dataFree()
{
	hipFree(d_clusters);
	hipFree(d_distances);
	hipFree(d_centers);
	hipFree(d_center_counts);
	hipFree(d_colors);
}

//A sz�ks�ges adatok visszam�sol�sa kernelr�l, majd a t�mb�k felszabad�t�sa
void slicCUDA::copyBackAndFree()
{
	hipMemcpy(distances, d_distances, sizeof(float)*rows*cols, hipMemcpyDeviceToHost);
	hipMemcpy(clusters, d_clusters, sizeof(int)*rows*cols, hipMemcpyDeviceToHost);
	hipMemcpy(centers, d_centers, sizeof(int)*centersLength * 5, hipMemcpyDeviceToHost);
	hipMemcpy(center_counts, d_center_counts, sizeof(int)*centersLength, hipMemcpyDeviceToHost);
	dataFree();
}

//A feldolgoz�s befejezt�vel az eredm�nyeket feldolgozva l�trehozok egy �j k�pet az eredm�nyek alapj�n
void slicCUDA::colour_with_cluster_means(Mat image) {
	for (int i = 0; i < image.cols; i++) {
		for (int j = 0; j < image.rows; j++) {
			//Kor�bban m�r meghat�roztam ,hogy az adott piyxelhez milyen sz�n tartozik, 
			//�gy csak be�ll�tom, hogy az �j k�pen is ez legyen a sz�ne.
			int idx = clusters[i*image.rows + j];
			Vec3b ncolour;

			ncolour.val[0] = centers[idx * 5 + 0];
			ncolour.val[1] = centers[idx * 5 + 1];
			ncolour.val[2] = centers[idx * 5 + 2];

			image.at<Vec3b>(j, i) = ncolour;
		}
	}
}

//El�sz�r kisz�molom, hogy h�ny blokkra lesz sz�ks�gem, majd elind�tom a megfelel� kerneleket
void slicCUDA::startKernels()
{
	int howManyBlocksInClusterProcess = centersLength / maxThreadinoneBlock;
	int threadsPerBlockInClusterProcess = (centersLength / howManyBlocksInClusterProcess) + 1;

	int howManyBlocksInPixelProcess = rows*cols / maxThreadinoneBlock;
	int threadsPerBlockInPixelProcess = (rows*cols / howManyBlocksInPixelProcess) + 1;

	for (int i = 0; i < iterations; i++)
	{
		dataCopy();
		orderingPixelsForClustersKernel << <howManyBlocksInClusterProcess, threadsPerBlockInClusterProcess >> >
			(cols, rows, step, centersLength, d_clusters, d_distances, d_centers, d_center_counts, d_colors, 5);
		clusterValuesSumByPixelsKernel << <howManyBlocksInPixelProcess, threadsPerBlockInPixelProcess >> >
			(cols, rows, d_clusters, d_distances, d_centers, d_center_counts, d_colors, 5);
		computeCorrectCentroidValuesKernel << <howManyBlocksInClusterProcess, threadsPerBlockInClusterProcess >> >
			(centersLength, d_centers, d_center_counts, 5);

		copyBackAndFree();
	}
}

//Itt ker�l tesztel�sre az, hogy a superpixelek j�l szegment�lt�k-e ki a k�pet.
//Ehhez az egyes szegmenseket nem az �tlagsz�nnel t�lt�m fel, hanem random sz�nekkel,
//annak �rdek�ben, hogy ez�ltal a szomsz�dos szegmensek j�l elk�l�n�lnek majd egym�st�l.
void slicCUDA::testSuperpixel(Mat image)
{
	for (int i = 0; i < centersLength; i++)
	{
		centers[i * 5 + 0] = rand() % 255 + 0;
		centers[i * 5 + 1] = rand() % 255 + 0;
		centers[i * 5 + 2] = rand() % 255 + 0;
	}

	for (int i = 0; i < image.cols; i++) {
		for (int j = 0; j < image.rows; j++) {
			int idx = clusters[i*image.rows + j];
			Vec3b ncolour;

			ncolour.val[0] = centers[idx * 5 + 0];
			ncolour.val[1] = centers[idx * 5 + 1];
			ncolour.val[2] = centers[idx * 5 + 2];

			image.at<Vec3b>(j, i) = ncolour;
		}
	}
	imwrite("testWithRandomColour.jpg", image);
}

//Itt ker�lnek ki�rat�sra a tesztel�s sor�n sz�ks�ges adatok a consolera
void slicCUDA::testDataToConsole()
{
	int howManyBlocksInClusterProcess = centersLength / maxThreadinoneBlock;
	int threadsPerBlockInClusterProcess = (centersLength / howManyBlocksInClusterProcess) + 1;

	int howManyBlocksInPixelProcess = rows*cols / maxThreadinoneBlock;
	int threadsPerBlockInPixelProcess = (rows*cols / howManyBlocksInPixelProcess) + 1;

	int notInCentroid = 0;
	for (int i = 0; i < rows*cols; i++)
	{
		if (clusters[i] == -1)
		{
			notInCentroid++;
		}
	}
	int inCentroid = rows*cols - notInCentroid;

	printf("%i a kep sorainak a szama\n", rows);
	printf("%i a kep oszlopainak a szama\n", cols);
	printf("%i darab pixelbol all �sszesen a kep\n", rows*cols);
	printf("%i tavolsagra kerultek elhelyezesre egymastol a centroidok\n\n", step);

	printf("%i darab centroidra van szukseg a feldoglozassoran\n", centersLength);
	printf("%i darab elinditott szal a clentroidok mopzgatasahoz.\n\n", threadsPerBlockInClusterProcess*howManyBlocksInClusterProcess);

	printf("%i darab elinditott szal a pixelek feldolgozasahoz.\n", howManyBlocksInPixelProcess*threadsPerBlockInPixelProcess);
	printf("%i darab pixel centroidhoz van renderve\n", inCentroid);
	printf("%i darab pixel nincs centroidhoz renderve\n", notInCentroid);

	getchar();
}
